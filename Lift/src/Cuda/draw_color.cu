#include "hip/hip_runtime.h"
#include "pch.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<float4, 2> result_buffer;

rtDeclareVariable(float3, draw_color, , );

RT_PROGRAMvoid draw_solid_color() {
	result_buffer[launch_index] = make_float4(draw_color, 0.f);
}
