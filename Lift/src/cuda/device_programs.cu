#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "launch_parameters.cuh"
#include "vector_functions.hpp"
#include "random.cuh"

namespace lift {
	extern "C" __constant__ LaunchParameters optix_launch_parameters;

	enum { SURFACE_RAY_TYPE = 0, RAY_TYPE_COUNT };

	static __forceinline__ __device__
	void* unpack_pointer(uint32_t i0, uint32_t i1) {
		const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
		void* ptr = reinterpret_cast<void*>(uptr);
		return ptr;
	}

	static __forceinline__ __device__
	void pack_pointer(void* ptr, uint32_t& i0, uint32_t& i1) {
		const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
		i0 = uptr >> 32;
		i1 = uptr & 0x00000000ffffffff;
	}

	template <typename T>
	static __forceinline__ __device__ T* get_payload() {
		const uint32_t u0 = optixGetPayload_0();
		const uint32_t u1 = optixGetPayload_1();
		return reinterpret_cast<T*>(unpack_pointer(u0, u1));
	}

	extern "C" __global__ void __closesthit__radiance() {
		//const int prim_id = optixGetPrimitiveIndex();
		//vec3& payload = *(vec3*)get_payload<vec3>();
		//payload = random_color(prim_id);
		const HitGroupData& sbt_data = *(const HitGroupData*)optixGetSbtDataPointer();

		// Compute normal
		const int prim_id = optixGetPrimitiveIndex();
		const ivec3* indices = reinterpret_cast<ivec3*>(sbt_data.geometry_data.triangle_mesh.indices.data);
		const ivec3 index = indices[prim_id];
		const vec3* positions = reinterpret_cast<vec3*>(sbt_data.geometry_data.triangle_mesh.positions.data);
		const vec3& vx = positions[index.x];
		const vec3& vy = positions[index.y];
		const vec3& vz = positions[index.z];
		const vec3 normal = normalize(cross(vy - vx, vz - vx));

		auto temp_dir = optixGetWorldRayDirection();
		const vec3 ray_dir = {
			temp_dir.x, temp_dir.y, temp_dir.z
		};

		const float cos_dn = 0.2f + 0.8f * fabsf(dot(ray_dir, normal));
		vec3& payload = *(vec3*)get_payload<vec3>();
		//payload = cos_dn * vec3(sbt_data.material_data.base_color);
		payload = vec3(1.0f, 0.4f, 0.4f);

	}

	extern "C" __global__ void __anyhit__radiance() {
		/*! for this simple example, this will remain empty */
	}

	extern "C" __global__ void __miss__radiance() {
		auto& payload = *static_cast<vec3*>(get_payload<vec3>());
		payload = vec3(1.0f);
	}

	extern "C" __global__ void __closesthit__occlusion() {
		/*! for this simple example, this will remain empty */

	}

	//------------------------------------------------------------------------------
	// ray gen program - the actual rendering happens in here
	//------------------------------------------------------------------------------
	extern "C" __global__ void __raygen__render_frame() {
		const int ix = optixGetLaunchIndex().x;
		const int iy = optixGetLaunchIndex().y;

		const auto& camera = optix_launch_parameters.camera;

		vec3 pixel_color_payload(0.0f);

		uint32_t u0, u1;
		pack_pointer(&pixel_color_payload, u0, u1);

		const vec2 screen(vec2(ix + 0.5f, iy + 0.5f) / vec2(optix_launch_parameters.frame.size));

		vec3 ray_dir = normalize(camera.direction +
			(screen.x - 0.5f) * camera.horizontal +
			(screen.y - 0.5f) * camera.vertical);

		optixTrace(optix_launch_parameters.traversable,
				   make_float3(camera.position.x, camera.position.y, camera.position.z),
				   make_float3(ray_dir.x, ray_dir.y, ray_dir.z),
				   0.0f,
				   1e20f,
				   0.0f,
				   OptixVisibilityMask(255),
				   OPTIX_RAY_FLAG_DISABLE_ANYHIT, //OPTIX_RAY_FLAG_NONE,
				   SURFACE_RAY_TYPE, // SBT offset
				   RAY_TYPE_COUNT, // SBT stride
				   SURFACE_RAY_TYPE, // missSBTIndex 
				   u0, u1);

		const int r = int(255.99f * pixel_color_payload.r);
		const int g = int(255.99f * pixel_color_payload.g);
		const int b = int(255.99f * pixel_color_payload.b);

		const uint32_t rgba = 0xff000000
			| (r << 0) | (g << 8) | (b << 16);
		//
		//
		const uint32_t fb_index = ix + iy * optix_launch_parameters.frame.size.x;
		optix_launch_parameters.frame.color_buffer[fb_index] = rgba;
	}
}
