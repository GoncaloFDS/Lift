#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "launch_parameters.h"
#include "vector_functions.hpp"
#include "random.cuh"
#include "vec_math.h"
#include "local_geometry.h"

namespace lift {
extern "C" __constant__ LaunchParameters params;

//------------------------------------------------------------------------------
//
// GGX/smith shading helpers
// TODO: move into header so can be shared by path tracer and bespoke renderers
//
//------------------------------------------------------------------------------

__device__ float3 schlick( const float3 spec_color, const float V_dot_H )
{
	return spec_color + ( make_float3( 1.0f ) - spec_color ) * powf( 1.0f - V_dot_H, 5.0f );
}


__device__ float vis( const float N_dot_L, const float N_dot_V, const float alpha )
{
	const float alpha_sq = alpha*alpha;

	const float ggx0 = N_dot_L * sqrtf( N_dot_V*N_dot_V * ( 1.0f - alpha_sq ) + alpha_sq );
	const float ggx1 = N_dot_V * sqrtf( N_dot_L*N_dot_L * ( 1.0f - alpha_sq ) + alpha_sq );

	return 2.0f * N_dot_L * N_dot_V / (ggx0+ggx1);
}


__device__ float ggxNormal( const float N_dot_H, const float alpha )
{
	const float alpha_sq   = alpha*alpha;
	const float N_dot_H_sq = N_dot_H*N_dot_H;
	const float x          = N_dot_H_sq*( alpha_sq - 1.0f ) + 1.0f;
	return alpha_sq/( M_PIf*x*x );
}


__device__ float3 linearize( float3 c )
{
	return make_float3(
		powf( c.x, 2.2f ),
		powf( c.y, 2.2f ),
		powf( c.z, 2.2f )
	);
}

//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------

static __forceinline__ __device__ void traceRadiance(
	OptixTraversableHandle handle,
	float3 ray_origin,
	float3 ray_direction,
	float tmin,
	float tmax,
	PayloadRadiance *payload
) {
	uint32_t u0 = 0, u1 = 0, u2 = 0, u3 = 0;
	optixTrace(
		handle,
		ray_origin, ray_direction,
		tmin,
		tmax,
		0.0f,                     // rayTime
		OptixVisibilityMask(1),
		OPTIX_RAY_FLAG_NONE,
		RAY_TYPE_RADIANCE,        // SBT offset
		RAY_TYPE_COUNT,           // SBT stride
		RAY_TYPE_RADIANCE,        // missSBTIndex
		u0, u1, u2, u3);

	payload->result.x = __int_as_float(u0);
	payload->result.y = __int_as_float(u1);
	payload->result.z = __int_as_float(u2);
	payload->depth = u3;
}

static __forceinline__ __device__ bool traceOcclusion(
	OptixTraversableHandle handle,
	float3                 ray_origin,
	float3                 ray_direction,
	float                  tmin,
	float                  tmax
)
{
	uint32_t occluded = 0u;
	optixTrace(
		handle,
		ray_origin,
		ray_direction,
		tmin,
		tmax,
		0.0f,                    // rayTime
		OptixVisibilityMask( 1 ),
		OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
		RAY_TYPE_OCCLUSION,      // SBT offset
		RAY_TYPE_COUNT,          // SBT stride
		RAY_TYPE_OCCLUSION,      // missSBTIndex
		occluded );
	return occluded;
}
__forceinline__ __device__ void setPayloadResult(float3 p) {
	optixSetPayload_0(float_as_int(p.x));
	optixSetPayload_1(float_as_int(p.y));
	optixSetPayload_2(float_as_int(p.z));
}

__forceinline__ __device__ void setPayloadOcclusion(bool occluded) {
	optixSetPayload_0(static_cast<uint32_t>( occluded ));
}

extern "C" __global__ void __closesthit__radiance() {
	const HitGroupData *hit_group_data = reinterpret_cast<HitGroupData *>(optixGetSbtDataPointer());
	const LocalGeometry geom = getLocalGeometry(hit_group_data->geometry_data);

	////
	// Retrieve material data
	//
	float3 base_color = make_float3(hit_group_data->material_data.base_color);
	if (hit_group_data->material_data.base_color_tex)
		base_color *= linearize(make_float3(
			tex2D<float4>(hit_group_data->material_data.base_color_tex, geom.UV.x, geom.UV.y)
		));

	float metallic = hit_group_data->material_data.metallic;
	float roughness = hit_group_data->material_data.roughness;
	float4 mr_tex = make_float4(1.0f);
	if (hit_group_data->material_data.metallic_roughness_tex)
		// MR tex is (occlusion, roughness, metallic )
		mr_tex = tex2D<float4>(hit_group_data->material_data.metallic_roughness_tex, geom.UV.x, geom.UV.y);
	roughness *= mr_tex.y;
	metallic *= mr_tex.z;


	//
	// Convert to material params
	//
	const float F0 = 0.04f;
	const float3 diff_color = base_color * (1.0f - F0) * (1.0f - metallic);
	const float3 spec_color = lerp(make_float3(F0), base_color, metallic);
	const float alpha = roughness * roughness;

	//
	// compute direct lighting
	//

	float3 N = geom.N;
	if (hit_group_data->material_data.normal_tex) {
		const float4 NN = 2.0f * tex2D<float4>(hit_group_data->material_data.normal_tex, geom.UV.x, geom.UV.y)
			- make_float4(1.0f);
		N = normalize(NN.x * normalize(geom.dpdu) + NN.y * normalize(geom.dpdv) + NN.z * geom.N);
	}

	float3 result = make_float3(0.0f);

	for (int i = 0; i < params.lights.count; ++i) {
		Light::Point light = params.lights[i];

		// TODO: optimize
		const float L_dist = length(light.position - geom.P);
		const float3 L = (light.position - geom.P) / L_dist;
		const float3 V = -normalize(optixGetWorldRayDirection());
		const float3 H = normalize(L + V);
		const float N_dot_L = dot(N, L);
		const float N_dot_V = dot(N, V);
		const float N_dot_H = dot(N, H);
		const float V_dot_H = dot(V, H);

		if (N_dot_L > 0.0f && N_dot_V > 0.0f) {
			const float tmin = 0.001f;          // TODO
			const float tmax = L_dist - 0.001f; // TODO
			const bool occluded = traceOcclusion(params.handle, geom.P, L, tmin, tmax);
			if (!occluded) {
				const float3 F = schlick(spec_color, V_dot_H);
				const float G_vis = vis(N_dot_L, N_dot_V, alpha);
				const float D = ggxNormal(N_dot_H, alpha);

				const float3 diff = (1.0f - F) * diff_color / M_PIf;
				const float3 spec = F * G_vis * D;

				result += light.color * light.intensity * N_dot_L * (diff + spec);
			}
		}
	}
	// TODO: add debug viewing mode that allows runtime switchable views of shading params, normals, etc
	//result = make_float3( roughness );
	//result = N*0.5f + make_float3( 0.5f );
	//result = geom.N*0.5f + make_float3( 0.5f );
	setPayloadResult(result);
}

extern "C" __global__ void __anyhit__radiance() {
	/*! for this simple example, this will remain empty */
}

extern "C" __global__ void __miss__radiance() {
	setPayloadResult(params.miss_color);
}

extern "C" __global__ void __closesthit__occlusion() {
	/*! for this simple example, this will remain empty */

}

//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__render_frame() {
	const uint3 launch_idx = optixGetLaunchIndex();
	const uint3 launch_dims = optixGetLaunchDimensions();
	const float3 eye = params.camera.eye;
	const float3 U = params.camera.U;
	const float3 V = params.camera.V;
	const float3 W = params.camera.W;
	const int subframe_index = params.subframe_index;

	//
	// Generate camera ray
	//
	uint32_t seed = tea<4>(launch_idx.y * launch_dims.x + launch_idx.x, subframe_index);
	const float2 subpixel_jitter = subframe_index == 0 ?
								   make_float2(0.0f, 0.0f) :
								   make_float2(rnd(seed) - 0.5f, rnd(seed) - 0.5f);

	const float2 d = 2.0f * make_float2(
		(static_cast<float>( launch_idx.x ) + subpixel_jitter.x) / static_cast<float>( launch_dims.x ),
		(static_cast<float>( launch_idx.y ) + subpixel_jitter.y) / static_cast<float>( launch_dims.y )
	) - 1.0f;
	const float3 ray_direction = normalize(d.x * U + d.y * V + W);
	const float3 ray_origin = eye;

	//
	// Trace camera ray
	//
	PayloadRadiance payload{
		make_float3(0.0f),
		1.0f,
		0
	};

	traceRadiance(
		params.handle,
		ray_origin,
		ray_direction,
		0.01f,
		1e16f,
		&payload
	);

	//
	// Update results
	//
	const uint32_t image_index = launch_idx.y * launch_dims.x + launch_idx.x;
	float3 accum_color = payload.result;

	if (subframe_index > 0) {
		const float a = 1.0f / static_cast<float>( subframe_index + 1 );
		const float3 accum_color_prev = make_float3(params.accum_buffer[image_index]);
		accum_color = lerp(accum_color_prev, accum_color, a);
	}
	params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
	params.frame_buffer[image_index] = make_color(accum_color);

}

}
