#include "hip/hip_runtime.h"
#include "optix_device.h"

#include "launch_parameters.cuh"
#include <crt/host_defines.h>

namespace lift {
	extern "C" __constant__ LaunchParameters optix_launch_parameters;

	extern "C" __global__ void __closesthit__radiance() {
		/*! for this simple example, this will remain empty */
	}

	extern "C" __global__ void __anyhit__radiance() {
		/*! for this simple example, this will remain empty */
	}

	extern "C" __global__ void __miss__radiance() {
		/*! for this simple example, this will remain empty */
	}


	//------------------------------------------------------------------------------
	// ray gen program - the actual rendering happens in here
	//------------------------------------------------------------------------------
	extern "C" __global__ void __raygen__render_frame() {
		if (optix_launch_parameters.frame_id == 0 &&
			optixGetLaunchIndex().x == 0 &&
			optixGetLaunchIndex().y == 0) {
			// we could of course also have used optixGetLaunchDims to query
			// the launch size, but accessing the optix_launch_parameters here
			// makes sure they're not getting optimized away (because
			// otherwise they'd not get used)
			printf("############################################\n");
			printf("Hello world from OptiX 7 raygen program!\n(within a %ix%i-sized launch)\n",
				   optix_launch_parameters.frame_buffer_size.x,
				   optix_launch_parameters.frame_buffer_size.y);
			printf("############################################\n");
		}

		// ------------------------------------------------------------------
		// for this example, produce a simple test pattern:
		// ------------------------------------------------------------------

		// compute a test pattern based on pixel ID
		const int ix = optixGetLaunchIndex().x;
		const int iy = optixGetLaunchIndex().y;

		const int r = (ix % 256);
		const int g = (iy % 256);
		const int b = ((ix + iy) % 256);

		// convert to 32-bit rgba value (we explicitly set alpha to 0xff
		// to make stb_image_write happy ...
		const uint32_t rgba = 0xff000000
			| (r << 0) | (g << 8) | (b << 16);

		// and write to frame buffer ...
		const uint32_t fbIndex = ix + iy * optix_launch_parameters.frame_buffer_size.x;
		optix_launch_parameters.color_buffer[fbIndex] = rgba;
	}
}
